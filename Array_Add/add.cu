#include <iostream>
#include <chrono>
#include <hip/hip_runtime.h>

// CUDA kernel for adding two arrays
__global__ void addArraysGPU(int* a, int* b, int* c, int n) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < n) {
        c[idx] = a[idx] + b[idx];
    }
}

// CPU function for adding two arrays
void addArraysCPU(int* a, int* b, int* c, int n) {
    for (int i = 0; i < n; i++) {
        c[i] = a[i] + b[i];
    }
}

int main() {
    const int N = 1 << 20; // Array size (1 million elements)
    const int size = N * sizeof(int);

    // Host arrays
    int *h_a = new int[N];
    int *h_b = new int[N];
    int *h_c_cpu = new int[N];
    int *h_c_gpu = new int[N];

    // Initialize arrays
    for (int i = 0; i < N; i++) {
        h_a[i] = i;
        h_b[i] = i * 2;
    }

    // CPU Timing
    auto start_cpu = std::chrono::high_resolution_clock::now();
    addArraysCPU(h_a, h_b, h_c_cpu, N);
    auto end_cpu = std::chrono::high_resolution_clock::now();
    double cpu_time = std::chrono::duration<double, std::milli>(end_cpu - start_cpu).count();

    // Allocate device memory
    int *d_a, *d_b, *d_c;
    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    // Copy data to device
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    // GPU Timing
    hipEvent_t start_gpu, end_gpu;
    hipEventCreate(&start_gpu);
    hipEventCreate(&end_gpu);

    hipEventRecord(start_gpu);
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    addArraysGPU<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, N);
    hipEventRecord(end_gpu);

    hipEventSynchronize(end_gpu);
    float gpu_time = 0;
    hipEventElapsedTime(&gpu_time, start_gpu, end_gpu);

    // Copy result back to host
    hipMemcpy(h_c_gpu, d_c, size, hipMemcpyDeviceToHost);

    // Validate GPU results
    bool isValid = true;
    for (int i = 0; i < N; i++) {
        if (h_c_cpu[i] != h_c_gpu[i]) {
            isValid = false;
            break;
        }
    }

    // Print results
    std::cout << "CPU Time: " << cpu_time << " ms\n";
    std::cout << "GPU Time: " << gpu_time << " ms\n";
    std::cout << "Time Difference: " << cpu_time - gpu_time << " ms\n";
    std::cout << "Results Valid: " << (isValid ? "Yes" : "No") << "\n";

    // Print some of the results to verify correctness
    std::cout << "\nFirst 10 results (CPU vs GPU):\n";
    for (int i = 0; i < 10; i++) {
        std::cout << "CPU: " << h_c_cpu[i] << " | GPU: " << h_c_gpu[i] << std::endl;
    }

    // Cleanup
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    delete[] h_a;
    delete[] h_b;
    delete[] h_c_cpu;
    delete[] h_c_gpu;

    return 0;
}
